#include "hip/hip_runtime.h"
#include <vector>
#include <type_traits>
#include <iterator>
#include <utility>

#include "gtest/gtest.h"
#include "helper_math.cuh"

#include "util_test.cuh"
#include "blockindexiter.cuh"

using namespace cbp;

// Compile time tests
TYPE_TRAIT_TEST(std::is_default_constructible, BlockIndexIterator);
TYPE_TRAIT_TEST(std::is_move_constructible, BlockIndexIterator);
TYPE_TRAIT_TEST(std::is_copy_constructible, BlockIndexIterator);
TYPE_TRAIT_TEST(std::is_copy_assignable, BlockIndexIterator);
TYPE_TRAIT_TEST(std::is_destructible, BlockIndexIterator);
static_assert(std::is_same<std::iterator_traits<BlockIndexIterator>::reference, const BlockIndex&>::value,
    "BlockIndexIterator reference type is not const BlockIndex&");

TEST(BlockIndexIteratorTest, Equality)
{
    const int3 volSize1 = make_int3(3), volSize2 = make_int3(5);
    const int3 blockSize1 = make_int3(2), blockSize2 = make_int3(4);
    EXPECT_EQ(BlockIndexIterator(), BlockIndexIterator());
    EXPECT_EQ(BlockIndexIterator(volSize1, blockSize1), BlockIndexIterator(volSize1, blockSize1));
    EXPECT_EQ(BlockIndexIterator(volSize1, blockSize1) + 1, BlockIndexIterator(volSize1, blockSize1) + 1);

    EXPECT_NE(BlockIndexIterator(volSize1, blockSize1), BlockIndexIterator(volSize2, blockSize1));
    EXPECT_NE(BlockIndexIterator(volSize1, blockSize1), BlockIndexIterator(volSize1, blockSize2));
    EXPECT_NE(BlockIndexIterator(volSize1, blockSize1), BlockIndexIterator(volSize1, blockSize1, blockSize1));
    EXPECT_NE(BlockIndexIterator(volSize1, blockSize1) + 1, BlockIndexIterator(volSize1, blockSize1));
}

TEST(BlockIndexIteratorTest, Swap)
{
    BlockIndexIterator a(make_int3(3), make_int3(2)), aOld(a);
    BlockIndexIterator b(make_int3(5), make_int3(4)), bOld(b);
    ASSERT_NE(a, b);
    ASSERT_NE(aOld, bOld);
    ASSERT_EQ(a, aOld);
    ASSERT_EQ(b, bOld);
    std::swap(a, b);
    EXPECT_NE(a, b);
    EXPECT_NE(a, aOld);
    EXPECT_NE(b, bOld);
    EXPECT_EQ(aOld, b);
    EXPECT_EQ(bOld, a);
}

TEST(BlockIndexIteratorTest, PseudoRandomInputIteratorCompliant)
{
    // See https://en.cppreference.com/w/cpp/named_req/InputIterator  and
    // https://en.cppreference.com/w/cpp/named_req/RandomAccessIterator for requirements
    BlockIndexIterator b(make_int3(10), make_int3(2));
    BlockIndexIterator bOrig = b;
    BlockIndexIterator expected;

    // Iterator requirements
    ASSERT_EQ(0, b.linearIndex());
    ASSERT_EQ(BlockIndex(0, 2), *b);
    ++b;
    EXPECT_EQ(1, b.linearIndex());
    EXPECT_EQ(BlockIndex(make_int3(2,0,0), make_int3(4,2,2)), *b);

    // InputIterator requirements
    expected = ++b;
    EXPECT_EQ((*b).startIdx, b->startIdx);
    EXPECT_EQ(expected, b);

    (void)b++; // Passes if this is compilable

    expected = b;
    ASSERT_EQ(expected, b);
    BlockIndex expectedBlockIndex = *b;
    BlockIndex actualBlockIndex = *b++;
    EXPECT_EQ(++expected, b);
    EXPECT_EQ(expectedBlockIndex, actualBlockIndex);

    // NOTE: from this point, only the requirements BlockIndexIterator actually conforms to are tested

    // ForwardIterator requirements
    expected = b;
    ASSERT_EQ(expected, b);
    EXPECT_EQ(++expected, ++b);
    BlockIndexIterator bOld = b++;
    EXPECT_EQ(expected, bOld);
    EXPECT_NE(bOld, b);

    expected = b;
    EXPECT_EQ(*expected, *b++);

    // BidirectionalIterator requirements
    expected = b;
    ASSERT_EQ(expected, b);
    EXPECT_EQ(--expected, --b);
    EXPECT_EQ(*expected, *b);

    expected = b;
    EXPECT_EQ(*expected, *b--);

    // RandomAccessIterator requirements
    expected = b;
    ASSERT_EQ(expected, b);
    ++(++(++expected)); // Move forward 3 times
    ASSERT_NE(expected, b);
    EXPECT_EQ(expected, b + 3);
    EXPECT_EQ(expected, 3 + b);
    EXPECT_EQ(expected, b += 3);
    EXPECT_EQ(expected, b);

    expected = b;
    ASSERT_EQ(expected, b);
    --(--(--expected)); // Move back 3 times
    ASSERT_NE(expected, b);
    EXPECT_EQ(expected, b - 3);
    EXPECT_EQ(expected, b -= 3);
    EXPECT_EQ(expected, b);

    b = bOrig + 3;;
    expected = bOrig + 2;
    EXPECT_EQ(expected, 5 - b);

    expected = b;
    ASSERT_EQ(expected, b);
    EXPECT_TRUE(expected <= b);
    EXPECT_TRUE(expected >= b);

    ++b;
    EXPECT_TRUE(expected <= b);
    EXPECT_TRUE(expected < b);

    b = expected;
    ++expected;
    EXPECT_TRUE(b <= expected);
    EXPECT_TRUE(b < expected);

    b = bOrig;
    EXPECT_EQ(*(b + 2), b[2]);

    b += 5;
    EXPECT_EQ(b - bOrig, 5);
    EXPECT_EQ(bOrig - b, -5);
}

// Helper function to build a BlockIndexIterator and extract its numBlocks field
inline int3 numBlocks(int3 volSize, int3 blockSize, int3 borderSize=make_int3(0))
{
    return BlockIndexIterator(volSize, blockSize, borderSize).numBlocks();
}

TEST(BlockIndexIteratorTest, CalcNumBlocks)
{
    EXPECT_EQ(make_int3(2), numBlocks(make_int3(4), make_int3(2)));
    EXPECT_EQ(make_int3(2), numBlocks(make_int3(3), make_int3(2)));
    EXPECT_EQ(make_int3(1), numBlocks(make_int3(2), make_int3(2)));
    EXPECT_EQ(make_int3(4, 2, 3), numBlocks(make_int3(8, 6, 12), make_int3(2, 3, 4)));
    EXPECT_EQ(make_int3(4, 2, 3), numBlocks(make_int3(7, 5, 11), make_int3(2, 3, 4)));
    EXPECT_EQ(make_int3(2), numBlocks(make_int3(4), make_int3(2), make_int3(2)));
    EXPECT_EQ(make_int3(2), numBlocks(make_int3(3), make_int3(2), make_int3(2)));
}

// Helper function to build a BlockIndexIterator and extract its blkIdx field
inline int maxBlkIdx(int3 volSize, int3 blockSize, int3 borderSize=make_int3(0))
{
    return BlockIndexIterator(volSize, blockSize, borderSize).maxLinearIndex();
}

TEST(BlockIndexIteratorTest, CalcMaxIdx)
{
    EXPECT_EQ(7, maxBlkIdx(make_int3(4), make_int3(2)));
    EXPECT_EQ(7, maxBlkIdx(make_int3(3), make_int3(2)));
    EXPECT_EQ(0, maxBlkIdx(make_int3(2), make_int3(2)));
    EXPECT_EQ(23, maxBlkIdx(make_int3(8, 6, 12), make_int3(2, 3, 4)));
    EXPECT_EQ(23, maxBlkIdx(make_int3(7, 5, 11), make_int3(2, 3, 4)));
    EXPECT_EQ(7, maxBlkIdx(make_int3(4), make_int3(2), make_int3(2)));
    EXPECT_EQ(7, maxBlkIdx(make_int3(3), make_int3(2), make_int3(2)));
}

TEST(BlockIndexIteratorTest, BeginEnd)
{
    const BlockIndexIterator bii1(make_int3(2), make_int3(1));
    EXPECT_EQ(0, bii1.linearIndex());
    EXPECT_EQ(0, bii1.begin().linearIndex());
    EXPECT_EQ(2*2*2, bii1.end().linearIndex());

    const BlockIndexIterator bii2(make_int3(8, 6, 12), make_int3(2, 3, 4));
    EXPECT_EQ(0, bii2.linearIndex());
    EXPECT_EQ(0, bii2.begin().linearIndex());
    EXPECT_EQ(4*2*3, bii2.end().linearIndex());
}

// Helper functions which check if block indices are computed correctly
void checkBlockIndices(const BlockIndexIterator& bii,
    std::vector<int> xIdxStart, std::vector<int> yIdxStart, std::vector<int> zIdxStart,
    std::vector<int> xIdxEnd, std::vector<int> yIdxEnd, std::vector<int> zIdxEnd,
    std::vector<int> xIdxStartBdr, std::vector<int> yIdxStartBdr, std::vector<int> zIdxStartBdr,
    std::vector<int> xIdxEndBdr, std::vector<int> yIdxEndBdr, std::vector<int> zIdxEndBdr)
{
    // X-lists
    ASSERT_EQ(xIdxStart.size(), xIdxEnd.size());
    ASSERT_EQ(xIdxStart.size(), xIdxStartBdr.size());
    ASSERT_EQ(xIdxStart.size(), xIdxEndBdr.size());

    // Y-lists
    ASSERT_EQ(yIdxStart.size(), yIdxEnd.size());
    ASSERT_EQ(yIdxStart.size(), yIdxStartBdr.size());
    ASSERT_EQ(yIdxStart.size(), yIdxEndBdr.size());

    // Z-lists
    ASSERT_EQ(zIdxStart.size(), zIdxEnd.size());
    ASSERT_EQ(zIdxStart.size(), zIdxStartBdr.size());
    ASSERT_EQ(zIdxStart.size(), zIdxEndBdr.size());

    int i = 0;
    for (int zi = 0; zi < zIdxStart.size(); zi++) {
        for (int yi = 0; yi < yIdxStart.size(); yi++) {
            for (int xi = 0; xi < xIdxStart.size(); xi++) {
                const BlockIndex b = bii.blockIndexAt(i);
                const int3 xyzIdx = make_int3(xi, yi, zi);
                const int3 startIdx = make_int3(xIdxStart[xi], yIdxStart[yi], zIdxStart[zi]);
                const int3 endIdx = make_int3(xIdxEnd[xi], yIdxEnd[yi], zIdxEnd[zi]);
                const int3 startIdxBdr = make_int3(xIdxStartBdr[xi], yIdxStartBdr[yi], zIdxStartBdr[zi]);
                const int3 endIdxBdr = make_int3(xIdxEndBdr[xi], yIdxEndBdr[yi], zIdxEndBdr[zi]);
                EXPECT_EQ(startIdx, b.startIdx) <<
                    "Wrong start index at " << xyzIdx << ", i=" << i;
                EXPECT_EQ(endIdx, b.endIdx) <<
                    "Wrong end index at " << xyzIdx << ", i=" << i;
                EXPECT_EQ(startIdxBdr, b.startIdxBorder) <<
                    "Wrong start index w/ border at " << xyzIdx << ", i=" << i;
                EXPECT_EQ(endIdxBdr, b.endIdxBorder) <<
                    "Wrong end index w/ border at " << xyzIdx << ", i=" << i;
                i++;
            }
        }
    }
}

void checkBlockIndices(const BlockIndexIterator& bii,
    std::vector<int> xIdxStart, std::vector<int> yIdxStart, std::vector<int> zIdxStart,
    std::vector<int> xIdxEnd, std::vector<int> yIdxEnd, std::vector<int> zIdxEnd)
{
    checkBlockIndices(bii,
        xIdxStart, yIdxStart, zIdxStart, xIdxEnd, yIdxEnd, zIdxEnd,
        xIdxStart, yIdxStart, zIdxStart, xIdxEnd, yIdxEnd, zIdxEnd);
}

TEST(BlockIndexIteratorTest, AlignedIndices)
{
    const int3 blockSize = make_int3(2, 3, 4);
    const int3 volSize = make_int3(6, 6, 8);
    const BlockIndexIterator bii(volSize, blockSize);

    std::vector<int> xStart = { 0, 2, 4 }, yStart = { 0, 3 }, zStart = { 0, 4 };
    std::vector<int> xEnd = { 2, 4, 6 }, yEnd = { 3, 6 }, zEnd = { 4, 8 };
    checkBlockIndices(bii, xStart, yStart, zStart, xEnd, yEnd, zEnd);
}

TEST(BlockIndexIteratorTest, UnalignedIndices)
{
    const int3 blockSize = make_int3(2, 3, 4);
    const int3 volSize = make_int3(5, 5, 6);
    const BlockIndexIterator bii(volSize, blockSize);

    std::vector<int> xStart = { 0, 2, 4 }, yStart = { 0, 3 }, zStart = { 0, 4 };
    std::vector<int> xEnd = { 2, 4, 5 }, yEnd = { 3, 5 }, zEnd = { 4, 6 };
    checkBlockIndices(bii, xStart, yStart, zStart, xEnd, yEnd, zEnd);
}

TEST(BlockIndexIteratorTest, IndicesWithBorders)
{
    const int3 blockSize = make_int3(2, 3, 4);
    const int3 borderSize = make_int3(2, 0, 3);
    const int3 volSize = make_int3(6, 6, 8);
    const BlockIndexIterator bii(volSize, blockSize, borderSize);

    std::vector<int> xStart = { 0, 2, 4 }, yStart = { 0, 3 }, zStart = { 0, 4 };
    std::vector<int> xStartBdr = { 0, 0, 2 }, yStartBdr = { 0, 3 }, zStartBdr = { 0, 1 };
    std::vector<int> xEnd = { 2, 4, 6 }, yEnd = { 3, 6 }, zEnd = { 4, 8 };
    std::vector<int> xEndBdr = { 4, 6, 6 }, yEndBdr = { 3, 6 }, zEndBdr = { 7, 8 };
    checkBlockIndices(bii, xStart, yStart, zStart, xEnd, yEnd, zEnd,
        xStartBdr, yStartBdr, zStartBdr, xEndBdr, yEndBdr, zEndBdr);
}

TEST(BlockIndexIteratorTest, Iteration)
{
    const int3 blockSize = make_int3(2, 3, 4);
    const int3 volSize = make_int3(6, 6, 8);
    const BlockIndexIterator bii(volSize, blockSize);
    const int NUM_ITERS = 3*2*2;

    BlockIndexIterator b = bii.begin();
    EXPECT_EQ(bii.begin(), b);
    for (int i = 0; i < NUM_ITERS; i++) {
        EXPECT_EQ(i, b.linearIndex());
        ++b;
    }
    EXPECT_EQ(bii.end(), b);
    ++b;
    EXPECT_EQ(bii.end(), b);

    int nIter = 0;
    for (auto b = bii.begin(); b != bii.end(); ++b) {
        EXPECT_EQ(nIter, b.linearIndex());
        nIter++;
        if (nIter > NUM_ITERS) {
            // Safe guard to ensure the test will finish
            nIter = -1;
            break;
        }
    }
    EXPECT_EQ(NUM_ITERS, nIter);

    nIter = 0;
    for (BlockIndex b : bii) {
        nIter++;
        if (nIter > NUM_ITERS) {
            // Safe guard to ensure the test will finish
            nIter = -1;
            break;
        }
    }
    EXPECT_EQ(NUM_ITERS, nIter);
}